/*
 * File: kernelsf.cu
 * License: Please see LICENSE file.
 * AccFFT: Massively Parallel FFT Library
 * Created by Amir Gholami on 11/23/2015
 * Email: contact@accfft.org
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ inline float testcase_gpu(float X,float Y,float Z){

  float sigma= 4;
  float pi=M_PI;
  float analytic;
  analytic=std::exp( -sigma * ( (X-pi)*(X-pi) + (Y-pi)*(Y-pi) + (Z-pi)*(Z-pi) ));
  if(analytic!=analytic) analytic=0;
  return analytic;
}
__global__ void initialize_gpu_kernel(float * a, int *n, int n2_, int* isize, int* istart ){
  unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int j = blockDim.y * blockIdx.y + threadIdx.y;
  unsigned int k = blockDim.z * blockIdx.z + threadIdx.z;

  if(i>=isize[0]) return;
  if(j>=isize[1]) return;
  if(k>=isize[2]) return;

  {
    float pi=M_PI;
    float X,Y,Z;
    long int ptr;

    X=2*pi/n[0]*(i+istart[0]);
    Y=2*pi/n[1]*(j+istart[1]);
    Z=2*pi/n[2]*k;

    ptr=i*isize[1]*n2_+j*n2_+k;
    a[ptr]=testcase_gpu(X,Y,Z);
  }
  return;

}// end initialize_gpu_kernel
void initialize_gpu(float *a,int*n, int * isize, int * istart) {

  int n2_=n[2]; // Outplace Transform requires no padding

  // corresponding GPU sizes
  int * n_gpu, *isize_gpu, *istart_gpu;
  hipMalloc((void**) &n_gpu,3*sizeof(int));
  hipMalloc((void**) &isize_gpu,3*sizeof(int));
  hipMalloc((void**) &istart_gpu,3*sizeof(int));

  // Copy the sizes to GPU
  hipMemcpy(n_gpu, n, 3*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(isize_gpu, isize, 3*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(istart_gpu, istart, 3*sizeof(int), hipMemcpyHostToDevice);



  int blocksInX = std::ceil(n[0]/4.);
  int blocksInY = std::ceil(n[1]/4.);
  int blocksInZ = std::ceil(n2_/4.);

  dim3 Dg(blocksInX, blocksInY, blocksInZ);
  dim3 Db(4, 4, 4);
  initialize_gpu_kernel<<<Dg, Db>>>(a,n_gpu,n2_,isize_gpu,istart_gpu);
  hipDeviceSynchronize();

  hipFree(n_gpu);
  hipFree(isize_gpu);
  hipFree(istart_gpu);
  return;
} // end initialize_gpu
